#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h> // Added for malloc/free

__global__ void vector_add(const float *A, const float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        // if this is over N we want to stop as thread is useless/empty and if it runs this it will crash
        C[i] = A[i] + B[i];
}

// How to split into different streams for your algorithm
int main()
{
    int N = 10000000;
    size_t size = N * sizeof(float);

    // Host memory pointers
    float *h_A = (float *)malloc(size); // Renamed A to h_A to avoid conflict with kernel argument
    float *h_B = (float *)malloc(size); // Renamed B to h_B to avoid conflict with kernel argument
    float *h_C = (float *)malloc(size); // Renamed C to h_C to avoid conflict with kernel argument

    for (int i = 0; i < N; i++)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Device memory pointers
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int half = N / 2;
    size_t half_size = size / 2;
    int threads = 256;
    int blocks_half = (half + threads - 1) / threads;

    // Stream 1: First half data transfer H->D and kernel launch
    hipMemcpyAsync(d_A, h_A, half_size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, h_B, half_size, hipMemcpyHostToDevice, stream1);
    vector_add<<<blocks_half, threads, 0, stream1>>>(d_A, d_B, d_C, half);
    hipMemcpyAsync(h_C, d_C, half_size, hipMemcpyDeviceToHost, stream1);

    // Stream 2: Second half data transfer H->D and kernel launch
    hipMemcpyAsync(d_A + half, h_A + half, half_size, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(d_B + half, h_B + half, half_size, hipMemcpyHostToDevice, stream2);
    vector_add<<<blocks_half, threads, 0, stream2>>>(d_A + half, d_B + half, d_C + half, half);
    hipMemcpyAsync(h_C + half, d_C + half, half_size, hipMemcpyDeviceToHost, stream2);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    printf("C[0] = %f, C[N-1] = %f\n", h_C[0], h_C[N - 1]);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}